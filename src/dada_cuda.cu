/***************************************************************************
 *  
 *    Copyright (C) 2011 by Andrew Jameson
 *    Licensed under the Academic Free License version 2.1
 * 
 ****************************************************************************/

#include "config.h"
#include "dada_cuda.h"
#include "sys/time.h"

/*! select the specified GPU as the active device */
int dada_cuda_select_device (int index)
{
  hipError_t error_id = hipSetDevice (index);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_select_device: hipSetDevice failed: %s\n",
                      hipGetErrorString(error_id));
    return -1;
  }
  return 0;
}

/*! get the number of CUDA devices */
int dada_cuda_get_device_count ()
{
  int device_count = 0;
  hipError_t error_id = hipGetDeviceCount(&device_count);
  if (error_id != hipSuccess) 
  {
    fprintf (stderr, "dada_cuda_get_device_count: hipGetDeviceCount failed: %s\n", 
	                   hipGetErrorString(error_id) );
    return -1;
  }
  return device_count;
}

/*! get the name of the specified CUDA device */
char * dada_cuda_get_device_name (int index)
{
  hipDeviceProp_t device_prop;
  hipError_t error_id = hipGetDeviceProperties(&device_prop, index);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_get_device_name: hipGetDeviceProperties failed: %s\n",
                     hipGetErrorString(error_id) );
    return 0;
  }

  return strdup(device_prop.name);
}

/*! register the data_block in the hdu via hipHostRegister */
int dada_cuda_dbregister (dada_hdu_t * hdu)
{
  ipcbuf_t * db = (ipcbuf_t *) hdu->data_block;

  // ensure that the data blocks are SHM locked
  if (ipcbuf_lock (db) < 0)
  {
    perror("dada_dbregister: ipcbuf_lock failed\n");
    return -1;
  }

  // dont register buffers if they reside on the device
  if (ipcbuf_get_device(db) >= 0)
    return 0;

  size_t bufsz = db->sync->bufsz;
  unsigned int flags = 0;
  hipError_t rval;

  // lock each data block buffer as cuda memory
  uint64_t ibuf;
  for (ibuf = 0; ibuf < db->sync->nbufs; ibuf++)
  {
    rval = hipHostRegister ((void *) db->buffer[ibuf], bufsz, flags);
    if (rval != hipSuccess)
    {
      perror("dada_dbregister:  hipHostRegister failed\n");
      return -1;
    }
  }
  
  return 0;
}

/*! unregister the data_block in the hdu via cudaHostUnRegister */
int dada_cuda_dbunregister (dada_hdu_t * hdu)
{
  ipcbuf_t * db = (ipcbuf_t *) hdu->data_block;
  hipError_t error_id;

  // dont unregister buffers if they reside on the device
  if (ipcbuf_get_device(db) >= 0)
    return 0;

  // lock each data block buffer as cuda memory
  uint64_t ibuf;
  for (ibuf = 0; ibuf < db->sync->nbufs; ibuf++)
  {
    error_id = hipHostUnregister ((void *) db->buffer[ibuf]);
    if (error_id != hipSuccess)
    {
      fprintf (stderr, "dada_dbunregister: hipHostUnregister failed: %s\n",
               hipGetErrorString(error_id));
      return -1;
    }
  }

  return 0;
}

/*! return a pointer to GPU device memory of bytes size */
void * dada_cuda_device_malloc ( size_t bytes)
{
  hipError_t error_id;
  void * device_memory;
  error_id = hipMalloc (&device_memory, bytes);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_device_malloc: could not allocate %ld bytes: %s\n", 
                      bytes, hipGetErrorString(error_id));
    return 0;
  }
  return device_memory;
}

/*! free the specified GPU device memory */
int dada_cuda_device_free (void * memory)
{
  hipError_t error_id;
  error_id = hipFree (memory);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_device_free: could not free memory: %s\n",
                      hipGetErrorString(error_id));
    return -1;
  }
  return 0;
}

/*! return a pointer to GPU host memory of bytes size */
void * dada_cuda_host_malloc (size_t bytes)
{
  hipError_t error_id;
  void * host_memory; 
  error_id = hipHostMalloc (&host_memory, bytes);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_host_malloc: could not allocate %ld bytes: %s\n",
                      bytes, hipGetErrorString(error_id));
    return 0;
  }
  return host_memory;
}

/*! free the specified GPU host memory */
int dada_cuda_host_free (void * memory)
{
  hipError_t error_id;
  error_id = hipHostFree (memory);
  if (error_id != hipSuccess)
  {
    fprintf (stderr, "dada_cuda_host_free: could not free memory: %s\n",
                      hipGetErrorString(error_id));
    return -1;
  }
  return 0;
}


/*! transfer the supplied buffer to the GPU */
float dada_cuda_device_transfer (void * from, void * to, size_t size, memory_mode_t mode, hipStream_t stream)
{
  hipError_t error_id;

  struct timeval start;
  struct timeval end;
  gettimeofday (&start, 0);

  if (mode == PINNED && stream != 0)
  {
    error_id = hipMemcpyAsync (to , from, size, hipMemcpyHostToDevice, stream);
    if (error_id != hipSuccess)
    {
      fprintf (stderr, "dada_cuda_device_transfer: hipMemcpyAsync (H2D) failed: %s\n",
               hipGetErrorString(error_id));
      return -1;
    }
    
    error_id = hipStreamSynchronize (stream);
    if (error_id != hipSuccess)    
    {    
      fprintf (stderr, "dada_cuda_device_transfer: hipStreamSynchronize failed: %s\n",
               hipGetErrorString(error_id));
      return -1;
    }
  }
  else
  {
    error_id = hipMemcpy (to, from, size, hipMemcpyHostToDevice);

    if (error_id != hipSuccess)
    {
      fprintf (stderr, "dada_cuda_device_transfer: memcpy failed: %s\n",
                        hipGetErrorString(error_id));
      return -1;
    }
    hipDeviceSynchronize();
  }
  gettimeofday (&end, 0);

  float elapsed = (float) ((end.tv_sec - start.tv_sec) * 1000) + 
                  (float) ((end.tv_usec - start.tv_usec) / 1000);

  return elapsed;
}

void check_error_stream (const char* method, hipStream_t stream)
{
  if (!stream)
  {
    fprintf (stderr, "called check_error_stream on invalid stream\n");
    exit (1);
  }
  else
  {
    hipStreamSynchronize (stream);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
      fprintf (stderr,  "method=%s, hipGetLastError=%s\n", method, hipGetErrorString (error));
      exit (1);
    }
  }
}

